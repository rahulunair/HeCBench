/* Copyright (c) 2014, NVIDIA CORPORATION. All rights reserved.
 *
   redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
 
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <random>
#include <chrono>
#include <algorithm>
#include <hip/hip_runtime.h>

#ifdef WITH_FULL_W_MATRIX
#define R_W_MATRICES_SMEM_SLOTS 15
#else
#define R_W_MATRICES_SMEM_SLOTS 12
#endif

#define CHECK_CUDA(call) do { \
  hipError_t status = call; \
  if( status != hipSuccess ) { \
    fprintf(stderr, "CUDA Error at line %d in %s: %s\n", __LINE__, __FILE__, hipGetErrorString(status)); \
    exit((int) status); \
  } \
} while(0)

#define HOST_DEVICE        __host__ __device__
#define HOST_DEVICE_INLINE __host__ __device__ __forceinline__

HOST_DEVICE_INLINE double3 operator+(const double3 &u, const double3 &v )
{
  return make_double3(u.x+v.x, u.y+v.y, u.z+v.z);
}

HOST_DEVICE_INLINE double4 operator+(const double4 &u, const double4 &v )
{
  return make_double4(u.x+v.x, u.y+v.y, u.z+v.z, u.w+v.w);
}

struct PayoffCall
{
  double m_K;
  HOST_DEVICE_INLINE PayoffCall(double K) : m_K(K) {}
  HOST_DEVICE_INLINE double operator()(double S) const { return fmax(S - m_K, 0.0); }
  HOST_DEVICE_INLINE int is_in_the_money(double S) const { return S > m_K; }
};

struct PayoffPut
{
  double m_K;
  HOST_DEVICE_INLINE PayoffPut(double K) : m_K(K) {}
  HOST_DEVICE_INLINE double operator()(double S) const { return fmax(m_K - S, 0.0); }
  HOST_DEVICE_INLINE int is_in_the_money(double S) const { return S < m_K; }
};


template< int NUM_THREADS_PER_BLOCK, typename Payoff >
__global__ __launch_bounds__(NUM_THREADS_PER_BLOCK)
void generate_paths_kernel(int num_timesteps, 
                           int num_paths, 
                           Payoff payoff,
                           double dt, 
                           double S0, 
                           double r, 
                           double sigma, 
                           const double *__restrict samples, 
                           double *__restrict paths)
{
  // The path generated by this thread.
  int path = blockIdx.x*NUM_THREADS_PER_BLOCK + threadIdx.x;

  // Early exit.
  if( path >= num_paths ) return;
  
  // Compute (r - sigma^2 / 2).
  const double r_min_half_sigma_sq_dt = (r - 0.5*sigma*sigma)*dt;
  // Compute sigma*sqrt(dt).
  const double sigma_sqrt_dt = sigma*sqrt(dt);

  // Keep the previous price.
  double S = S0;

  // The offset.
  int offset = path;
  
  // Each thread generates several timesteps. 
  for( int timestep = 0 ; timestep < num_timesteps-1 ; ++timestep, offset += num_paths )
  {
    S = S * exp(r_min_half_sigma_sq_dt + sigma_sqrt_dt*samples[offset]);
    paths[offset] = S;
  }

  // The asset price.
  S = S * exp(r_min_half_sigma_sq_dt + sigma_sqrt_dt*samples[offset]);

  // Store the payoff at expiry.
  paths[offset] = payoff(S);
}

static __device__ __forceinline__ void assemble_R(int m, double4 &sums, double *smem_svds)
{
  // Assemble R.

  double x0 = smem_svds[0];
  double x1 = smem_svds[1];
  double x2 = smem_svds[2];

  double x0_sq = x0 * x0;

  double sum1 = sums.x - x0;
  double sum2 = sums.y - x0_sq;
  double sum3 = sums.z - x0_sq*x0;
  double sum4 = sums.w - x0_sq*x0_sq;

  double m_as_dbl = (double) m;
  double sigma = m_as_dbl - 1.0;
  double mu = sqrt(m_as_dbl);
  double v0 = -sigma / (1.0 + mu);
  double v0_sq = v0*v0;
  double beta = 2.0 * v0_sq / (sigma + v0_sq);
  
  double inv_v0 = 1.0 / v0;
  double one_min_beta = 1.0 - beta;
  double beta_div_v0  = beta * inv_v0;
  
  smem_svds[0] = mu;
  smem_svds[1] = one_min_beta*x0 - beta_div_v0*sum1;
  smem_svds[2] = one_min_beta*x0_sq - beta_div_v0*sum2;
  
  // Rank update coefficients.
  
  double beta_div_v0_sq = beta_div_v0 * inv_v0;
  
  double c1 = beta_div_v0_sq*sum1 + beta_div_v0*x0;
  double c2 = beta_div_v0_sq*sum2 + beta_div_v0*x0_sq;

  // 2nd step of QR.
  
  double x1_sq = x1*x1;

  sum1 -= x1;
  sum2 -= x1_sq;
  sum3 -= x1_sq*x1;
  sum4 -= x1_sq*x1_sq;
  
  x0 = x1-c1;
  x0_sq = x0*x0;
  sigma = sum2 - 2.0*c1*sum1 + (m_as_dbl-2.0)*c1*c1;
  if( abs(sigma) < 1.0e-16 )
    beta = 0.0;
  else
  {
    mu = sqrt(x0_sq + sigma);
    if( x0 <= 0.0 )
      v0 = x0 - mu;
    else
      v0 = -sigma / (x0 + mu);
    v0_sq = v0*v0;
    beta = 2.0*v0_sq / (sigma + v0_sq);
  }
  
  inv_v0 = 1.0 / v0;
  beta_div_v0 = beta * inv_v0;
  
  // The coefficient to perform the rank update.
  double c3 = (sum3 - c1*sum2 - c2*sum1 + (m_as_dbl-2.0)*c1*c2)*beta_div_v0;
  double c4 = (x1_sq-c2)*beta_div_v0 + c3*inv_v0;
  double c5 = c1*c4 - c2;
  
  one_min_beta = 1.0 - beta;
  
  // Update R. 
  smem_svds[3] = one_min_beta*x0 - beta_div_v0*sigma;
  smem_svds[4] = one_min_beta*(x1_sq-c2) - c3;
  
  // 3rd step of QR.
  
  double x2_sq = x2*x2;

  sum1 -= x2;
  sum2 -= x2_sq;
  sum3 -= x2_sq*x2;
  sum4 -= x2_sq*x2_sq;
  
  x0 = x2_sq-c4*x2+c5;
  sigma = sum4 - 2.0*c4*sum3 + (c4*c4 + 2.0*c5)*sum2 - 2.0*c4*c5*sum1 + (m_as_dbl-3.0)*c5*c5;
  if( abs(sigma) < 1.0e-12 )
    beta = 0.0;
  else
  {
    mu = sqrt(x0*x0 + sigma);
    if( x0 <= 0.0 )
      v0 = x0 - mu;
    else
      v0 = -sigma / (x0 + mu);
    v0_sq = v0*v0;
    beta = 2.0*v0_sq / (sigma + v0_sq);
  }
  
  // Update R.
  smem_svds[5] = (1.0-beta)*x0 - (beta/v0)*sigma;
}

static __host__ __device__ double off_diag_norm(double A01, double A02, double A12)
{
  return sqrt(2.0 * (A01*A01 + A02*A02 + A12*A12));
}

static __device__ __forceinline__ void swap(double &x, double &y)
{
  double t = x; x = y; y = t;
}

static __device__ __forceinline__ void svd_3x3(int m, double4 &sums, double *smem_svds)
{
  // Assemble the R matrix.
  assemble_R(m, sums, smem_svds);

  // The matrix R.
  double R00 = smem_svds[0];
  double R01 = smem_svds[1];
  double R02 = smem_svds[2];
  double R11 = smem_svds[3];
  double R12 = smem_svds[4];
  double R22 = smem_svds[5];

  // We compute the eigenvalues/eigenvectors of A = R^T R.
  
  double A00 = R00*R00;
  double A01 = R00*R01;
  double A02 = R00*R02;
  double A11 = R01*R01 + R11*R11;
  double A12 = R01*R02 + R11*R12;
  double A22 = R02*R02 + R12*R12 + R22*R22;
  
  // We keep track of V since A = Sigma^2 V. Each thread stores a row of V.
  
  double V00 = 1.0, V01 = 0.0, V02 = 0.0;
  double V10 = 0.0, V11 = 1.0, V12 = 0.0;
  double V20 = 0.0, V21 = 0.0, V22 = 1.0;
  
  // The Jacobi algorithm is iterative. We fix the max number of iter and the minimum tolerance.
  
  const int max_iters = 16;
  const double tolerance = 1.0e-12;
  
  // Iterate until we reach the max number of iters or the tolerance.
 
  for( int iter = 0 ; off_diag_norm(A01, A02, A12) >= tolerance && iter < max_iters ; ++iter )
  {
    double c, s, B00, B01, B02, B10, B11, B12, B20, B21, B22;
    
    // Compute the Jacobi matrix for p=0 and q=1.
    
    c = 1.0, s = 0.0;
    if( A01 != 0.0 )
    {
      double tau = (A11 - A00) / (2.0 * A01);
      double sgn = tau < 0.0 ? -1.0 : 1.0;
      double t   = sgn / (sgn*tau + sqrt(1.0 + tau*tau));
      
      c = 1.0 / sqrt(1.0 + t*t);
      s = t*c;
    }
    
    // Update A = J^T A J and V = V J.
    
    B00 = c*A00 - s*A01;
    B01 = s*A00 + c*A01;
    B10 = c*A01 - s*A11;
    B11 = s*A01 + c*A11;
    B02 = A02;
    
    A00 = c*B00 - s*B10;
    A01 = c*B01 - s*B11;
    A11 = s*B01 + c*B11;
    A02 = c*B02 - s*A12;
    A12 = s*B02 + c*A12;
    
    B00 = c*V00 - s*V01;
    V01 = s*V00 + c*V01;
    V00 = B00;
    
    B10 = c*V10 - s*V11;
    V11 = s*V10 + c*V11;
    V10 = B10;
    
    B20 = c*V20 - s*V21;
    V21 = s*V20 + c*V21;
    V20 = B20;
    
    // Compute the Jacobi matrix for p=0 and q=2.
    
    c = 1.0, s = 0.0;
    if( A02 != 0.0 )
    {
      double tau = (A22 - A00) / (2.0 * A02);
      double sgn = tau < 0.0 ? -1.0 : 1.0;
      double t   = sgn / (sgn*tau + sqrt(1.0 + tau*tau));
      
      c = 1.0 / sqrt(1.0 + t*t);
      s = t*c;
    }
    
    // Update A = J^T A J and V = V J.
    
    B00 = c*A00 - s*A02;
    B01 = c*A01 - s*A12;
    B02 = s*A00 + c*A02;
    B20 = c*A02 - s*A22;
    B22 = s*A02 + c*A22;
    
    A00 = c*B00 - s*B20;
    A12 = s*A01 + c*A12;
    A02 = c*B02 - s*B22;
    A22 = s*B02 + c*B22;
    A01 = B01;
    
    B00 = c*V00 - s*V02;
    V02 = s*V00 + c*V02;
    V00 = B00;
    
    B10 = c*V10 - s*V12;
    V12 = s*V10 + c*V12;
    V10 = B10;
    
    B20 = c*V20 - s*V22;
    V22 = s*V20 + c*V22;
    V20 = B20;
    
    // Compute the Jacobi matrix for p=1 and q=2.
    
    c = 1.0, s = 0.0;
    if( A12 != 0.0 )
    {
      double tau = (A22 - A11) / (2.0 * A12);
      double sgn = tau < 0.0 ? -1.0 : 1.0;
      double t   = sgn / (sgn*tau + sqrt(1.0 + tau*tau));
      
      c = 1.0 / sqrt(1.0 + t*t);
      s = t*c;
    }
    
    // Update A = J^T A J and V = V J.
    
    B02 = s*A01 + c*A02;
    B11 = c*A11 - s*A12;
    B12 = s*A11 + c*A12;
    B21 = c*A12 - s*A22;
    B22 = s*A12 + c*A22;
    
    A01 = c*A01 - s*A02;
    A02 = B02;
    A11 = c*B11 - s*B21;
    A12 = c*B12 - s*B22;
    A22 = s*B12 + c*B22;
    
    B01 = c*V01 - s*V02;
    V02 = s*V01 + c*V02;
    V01 = B01;
    
    B11 = c*V11 - s*V12;
    V12 = s*V11 + c*V12;
    V11 = B11;
    
    B21 = c*V21 - s*V22;
    V22 = s*V21 + c*V22;
    V21 = B21;
  }

  // Swap the columns to have S[0] >= S[1] >= S[2].
  if( A00 < A11 )
  {
    swap(A00, A11);
    swap(V00, V01);
    swap(V10, V11);
    swap(V20, V21);
  }
  if( A00 < A22 )
  {
    swap(A00, A22);
    swap(V00, V02);
    swap(V10, V12);
    swap(V20, V22);
  }
  if( A11 < A22 )
  {
    swap(A11, A22);
    swap(V01, V02);
    swap(V11, V12);
    swap(V21, V22);
  }

  //printf("timestep=%3d, svd0=%.8lf svd1=%.8lf svd2=%.8lf\n", blockIdx.x, sqrt(A00), sqrt(A11), sqrt(A22));
  
  // Invert the diagonal terms and compute V*S^-1.
  
  double inv_S0 = abs(A00) < 1.0e-12 ? 0.0 : 1.0 / A00;
  double inv_S1 = abs(A11) < 1.0e-12 ? 0.0 : 1.0 / A11;
  double inv_S2 = abs(A22) < 1.0e-12 ? 0.0 : 1.0 / A22;

  // printf("SVD: timestep=%3d %12.8lf %12.8lf %12.8lf\n", blockIdx.x, sqrt(A00), sqrt(A11), sqrt(A22));
  
  double U00 = V00 * inv_S0; 
  double U01 = V01 * inv_S1; 
  double U02 = V02 * inv_S2;
  double U10 = V10 * inv_S0; 
  double U11 = V11 * inv_S1; 
  double U12 = V12 * inv_S2;
  double U20 = V20 * inv_S0; 
  double U21 = V21 * inv_S1; 
  double U22 = V22 * inv_S2;
  
  // Compute V*S^-1*V^T*R^T.
  
#ifdef WITH_FULL_W_MATRIX
  double B00 = U00*V00 + U01*V01 + U02*V02;
  double B01 = U00*V10 + U01*V11 + U02*V12;
  double B02 = U00*V20 + U01*V21 + U02*V22;
  double B10 = U10*V00 + U11*V01 + U12*V02;
  double B11 = U10*V10 + U11*V11 + U12*V12;
  double B12 = U10*V20 + U11*V21 + U12*V22;
  double B20 = U20*V00 + U21*V01 + U22*V02;
  double B21 = U20*V10 + U21*V11 + U22*V12;
  double B22 = U20*V20 + U21*V21 + U22*V22;
  
  smem_svds[ 6] = B00*R00 + B01*R01 + B02*R02;
  smem_svds[ 7] =           B01*R11 + B02*R12;
  smem_svds[ 8] =                     B02*R22;
  smem_svds[ 9] = B10*R00 + B11*R01 + B12*R02;
  smem_svds[10] =           B11*R11 + B12*R12;
  smem_svds[11] =                     B12*R22;
  smem_svds[12] = B20*R00 + B21*R01 + B22*R02;
  smem_svds[13] =           B21*R11 + B22*R12;
  smem_svds[14] =                     B22*R22;
#else
  double B00 = U00*V00 + U01*V01 + U02*V02;
  double B01 = U00*V10 + U01*V11 + U02*V12;
  double B02 = U00*V20 + U01*V21 + U02*V22;
  double B11 = U10*V10 + U11*V11 + U12*V12;
  double B12 = U10*V20 + U11*V21 + U12*V22;
  double B22 = U20*V20 + U21*V21 + U22*V22;
  
  smem_svds[ 6] = B00*R00 + B01*R01 + B02*R02;
  smem_svds[ 7] =           B01*R11 + B02*R12;
  smem_svds[ 8] =                     B02*R22;
  smem_svds[ 9] =           B11*R11 + B12*R12;
  smem_svds[10] =                     B12*R22;
  smem_svds[11] =                     B22*R22;
#endif
}

template< int NUM_THREADS_PER_BLOCK, typename Payoff >
__global__ __launch_bounds__(NUM_THREADS_PER_BLOCK, 4)
void prepare_svd_kernel(int num_paths, 
                        int min_in_the_money, 
                        Payoff payoff, 
                        const double *__restrict paths, 
                                 int *__restrict all_out_of_the_money, 
                              double *__restrict svds)
{
  // We need to perform a scan to find the first 3 stocks pay off.
  __shared__ int scan_input[NUM_THREADS_PER_BLOCK];
  __shared__ int scan_output[1+NUM_THREADS_PER_BLOCK];

  // sum reduction
  __shared__ double4 lsums;
  __shared__ int lsum;

  // Shared buffer for the ouput.
  __shared__ double smem_svds[R_W_MATRICES_SMEM_SLOTS];

  // Each block works on a single timestep. 
  const int timestep = blockIdx.x;
  // The timestep offset.
  const int offset = timestep * num_paths;

  // Sums.
  int m = 0; double4 sums = { 0.0, 0.0, 0.0, 0.0 };

  // Initialize the shared memory. DBL_MAX is a marker to specify that the value is invalid.
  if( threadIdx.x < R_W_MATRICES_SMEM_SLOTS )
    smem_svds[threadIdx.x] = 0.0;
  __syncthreads();

  // Have we already found our 3 first paths which pay off.
  int found_paths = 0;

  // Iterate over the paths.
  //for( int path = threadIdx.x ; path < num_paths ; path += NUM_THREADS_PER_BLOCK )
  for( int path = threadIdx.x ; path < num_paths ; path += NUM_THREADS_PER_BLOCK )
  {
    // Load the asset price to determine if it pays off.
    double S = 0.0;
    if( path < num_paths )
      S = paths[offset + path];

    // Check if it pays off.
    const int in_the_money = payoff.is_in_the_money(S);

    // Try to check if we have found the 3 first stocks.
    scan_input[threadIdx.x] = in_the_money;
    __syncthreads();
    if (threadIdx.x == 0) {
      scan_output[0] = 0;
      for (int i = 1; i <= NUM_THREADS_PER_BLOCK; i++) 
        scan_output[i] = scan_output[i-1]+scan_input[i-1];
    }
    __syncthreads();
    const int partial_sum = scan_output[threadIdx.x];
    const int total_sum = scan_output[NUM_THREADS_PER_BLOCK];

    if( found_paths < 3 )
    {
      if( in_the_money && found_paths + partial_sum < 3 )
        smem_svds[found_paths + partial_sum] = S;
      __syncthreads();
      found_paths += total_sum;
    }

    // Early continue if no item pays off.
    if (threadIdx.x == 0) lsum = 0;
    __syncthreads();
    atomicOr(&lsum, in_the_money);
    __syncthreads();
    if (lsum == 0) continue;
    
    // Update the number of payoff items.
    m += in_the_money;

    // The "normalized" value.
    double x = 0.0, x_sq = 0.0;
    if( in_the_money )
    {
      x = S;
      x_sq = S*S;
    }

    // Compute the 4 sums.
    sums.x += x;
    sums.y += x_sq;
    sums.z += x_sq*x;
    sums.w += x_sq*x_sq;
  }

  // Compute the final reductions.
  if (threadIdx.x == 0) lsum = 0;
  __syncthreads();

  atomicAdd(&lsum, m);

  __syncthreads();

  int not_enough_paths = 0;
  // Do we all exit?
  if (threadIdx.x == 0 && lsum < min_in_the_money)
    not_enough_paths = 1;
  
  // Early exit if no path is in the money.
  if( not_enough_paths )
  {
    if( threadIdx.x == 0 )
      all_out_of_the_money[blockIdx.x] = 1;
  } 
  else
  {
    // Compute the final reductions.

    if (threadIdx.x == 0)
      lsums = make_double4(0,0,0,0);
    __syncthreads();

    atomicAdd(&lsums.x, sums.x);
    atomicAdd(&lsums.y, sums.y);
    atomicAdd(&lsums.z, sums.z);
    atomicAdd(&lsums.w, sums.w);
    
    __syncthreads();
    
    // The 1st thread has everything he needs to build R from the QR decomposition.
    if( threadIdx.x == 0 )
      svd_3x3(lsum, lsums, smem_svds);

    __syncthreads();

    // Store the final results.
    if( threadIdx.x < R_W_MATRICES_SMEM_SLOTS )
      svds[16*blockIdx.x + threadIdx.x] = smem_svds[threadIdx.x];
  }
}

template< int NUM_THREADS_PER_BLOCK, typename Payoff >
__global__ __launch_bounds__(NUM_THREADS_PER_BLOCK, 8)
void compute_partial_beta_kernel(int num_paths,
                                 Payoff payoff,
                                 const double *__restrict svd,
                                 const double *__restrict paths,
                                 const double *__restrict cashflows,
                                 const int *__restrict all_out_of_the_money,
                                 double *__restrict partial_sums)
{
  // The shared memory storage.
  __shared__ double3 lsums;
  
  // The shared memory to store the SVD.
  __shared__ double shared_svd[R_W_MATRICES_SMEM_SLOTS];
    
  // Early exit if needed.
  if( *all_out_of_the_money ) return;

  // The number of threads per grid.
  const int NUM_THREADS_PER_GRID = NUM_THREADS_PER_BLOCK * gridDim.x;

  // The 1st threads loads the matrices SVD and R.
  if( threadIdx.x < R_W_MATRICES_SMEM_SLOTS )
    shared_svd[threadIdx.x] = svd[threadIdx.x];
  __syncthreads();

  // Load the terms of R.
  const double R00 = shared_svd[ 0];
  const double R01 = shared_svd[ 1];
  const double R02 = shared_svd[ 2];
  const double R11 = shared_svd[ 3];
  const double R12 = shared_svd[ 4];
  const double R22 = shared_svd[ 5];

  // Load the elements of W.
#ifdef WITH_FULL_W_MATRIX
  const double W00 = shared_svd[ 6];
  const double W01 = shared_svd[ 7];
  const double W02 = shared_svd[ 8];
  const double W10 = shared_svd[ 9];
  const double W11 = shared_svd[10];
  const double W12 = shared_svd[11];
  const double W20 = shared_svd[12];
  const double W21 = shared_svd[13];
  const double W22 = shared_svd[14];
#else
  const double W00 = shared_svd[ 6];
  const double W01 = shared_svd[ 7];
  const double W02 = shared_svd[ 8];
  const double W11 = shared_svd[ 9];
  const double W12 = shared_svd[10];
  const double W22 = shared_svd[11];
#endif

  // Invert the diagonal of R.
  const double inv_R00 = R00 != 0.0 ? __drcp_rn(R00) : 0.0;
  const double inv_R11 = R11 != 0.0 ? __drcp_rn(R11) : 0.0;
  const double inv_R22 = R22 != 0.0 ? __drcp_rn(R22) : 0.0;

  // Precompute the R terms.
  const double inv_R01 = inv_R00*inv_R11*R01;
  const double inv_R02 = inv_R00*inv_R22*R02;
  const double inv_R12 =         inv_R22*R12;
  
  // Precompute W00/R00.
#ifdef WITH_FULL_W_MATRIX
  const double inv_W00 = W00*inv_R00;
  const double inv_W10 = W10*inv_R00;
  const double inv_W20 = W20*inv_R00;
#else
  const double inv_W00 = W00*inv_R00;
#endif

  // Each thread has 3 numbers to sum.
  double beta0 = 0.0, beta1 = 0.0, beta2 = 0.0;

  // Iterate over the paths.
  for( int path = blockIdx.x*NUM_THREADS_PER_BLOCK + threadIdx.x ; path < num_paths ; path += NUM_THREADS_PER_GRID )
  {
    // Threads load the asset price to rebuild Q from the QR decomposition.
    double S = paths[path];

    // Is the path in the money?
    const int in_the_money = payoff.is_in_the_money(S);

    // Compute Qis. The elements of the Q matrix in the QR decomposition.
    double Q1i = inv_R11*S - inv_R01;
    double Q2i = inv_R22*S*S - inv_R02 - Q1i*inv_R12;

    // Compute the ith row of the pseudo-inverse of [1 X X^2].
#ifdef WITH_FULL_W_MATRIX
    const double WI0 = inv_W00 + W01 * Q1i + W02 * Q2i;
    const double WI1 = inv_W10 + W11 * Q1i + W12 * Q2i;
    const double WI2 = inv_W20 + W21 * Q1i + W22 * Q2i;
#else
    const double WI0 = inv_W00 + W01 * Q1i + W02 * Q2i;
    const double WI1 =           W11 * Q1i + W12 * Q2i;
    const double WI2 =                       W22 * Q2i;
#endif

    // Each thread loads its element from the Y vector.
    double cashflow = in_the_money ? cashflows[path] : 0.0;
  
    // Update beta.
    beta0 += WI0*cashflow;
    beta1 += WI1*cashflow;
    beta2 += WI2*cashflow;
  }

  // Compute the sum of the elements in the block. 
  if( threadIdx.x == 0 )
    lsums = make_double3(0,0,0);
  __syncthreads();

  atomicAdd(&lsums.x, beta0);
  atomicAdd(&lsums.y, beta1);
  atomicAdd(&lsums.z, beta2);
 
  __syncthreads();
  
  // The 1st thread stores the result to GMEM.
  if( threadIdx.x == 0 )
  {
    partial_sums[0*NUM_THREADS_PER_BLOCK + blockIdx.x] = lsums.x;
    partial_sums[1*NUM_THREADS_PER_BLOCK + blockIdx.x] = lsums.y;
    partial_sums[2*NUM_THREADS_PER_BLOCK + blockIdx.x] = lsums.z;
  }
}

template< int NUM_THREADS_PER_BLOCK >
__global__ __launch_bounds__(NUM_THREADS_PER_BLOCK)
void compute_final_beta_kernel(const int *__restrict all_out_of_the_money, double *__restrict beta)
{
  // The shared memory for the reduction.
  __shared__ double3 lsums;

  // Early exit if needed.
  if( *all_out_of_the_money )
  {
    if( threadIdx.x < 3 )
      beta[threadIdx.x] = 0.0;
    return;
  }

  // The final sums.
  double3 sums;
  
  // We load the elements.
  sums.x = beta[0*NUM_THREADS_PER_BLOCK + threadIdx.x];
  sums.y = beta[1*NUM_THREADS_PER_BLOCK + threadIdx.x];
  sums.z = beta[2*NUM_THREADS_PER_BLOCK + threadIdx.x];
  
  // Compute the sums.
  if( threadIdx.x == 0 )
    lsums = make_double3(0,0,0);
  __syncthreads();

  atomicAdd(&lsums.x, sums.x);
  atomicAdd(&lsums.y, sums.y);
  atomicAdd(&lsums.z, sums.z);
 
  __syncthreads();
  
  // Store beta.
  if( threadIdx.x == 0 )
  {
    //printf("beta0=%.8lf beta1=%.8lf beta2=%.8lf\n", sums.x, sums.y, sums.z);
    beta[0] = lsums.x; 
    beta[1] = lsums.y;
    beta[2] = lsums.z;
  }
}

// assumes beta has been built either by compute_final_beta_kernel or
// by atomic operations at the end of compute_partial_beta_kernel.

template< int NUM_THREADS_PER_BLOCK, typename Payoff >
__global__ __launch_bounds__(NUM_THREADS_PER_BLOCK)
void update_cashflow_kernel(int num_paths,
                            Payoff payoff_object,
                            double exp_min_r_dt,
                            const double *__restrict beta,
                            const double *__restrict paths,
                            const int *__restrict all_out_of_the_money,
                            double *__restrict cashflows)
{
  const int NUM_THREADS_PER_GRID = gridDim.x * NUM_THREADS_PER_BLOCK;

  // Are we going to skip the computations.
  const int skip_computations = *all_out_of_the_money;

  // Load the beta coefficients for the linear regression.
  const double beta0 = beta[0];
  const double beta1 = beta[1];
  const double beta2 = beta[2];

  // Iterate over the paths.
  int path = blockIdx.x*NUM_THREADS_PER_BLOCK + threadIdx.x;
  for( ; path < num_paths ; path += NUM_THREADS_PER_GRID )
  {
    // The cashflow.
    const double old_cashflow = exp_min_r_dt*cashflows[path];
    if( skip_computations )
    {
      cashflows[path] = old_cashflow;
      continue;
    }
  
    // Load the asset price.
    double S  = paths[path];
    double S2 = S*S;

    // The payoff.
    double payoff = payoff_object(S);

    // Compute the estimated payoff from continuing.
    double estimated_payoff = beta0 + beta1*S + beta2*S2;

    // Discount the payoff because we did not take it into account for beta.
    estimated_payoff *= exp_min_r_dt;

    // Update the payoff.
    if( payoff <= 1.0e-8 || payoff <= estimated_payoff )
      payoff = old_cashflow;
    
    // Store the updated cashflow.
    cashflows[path] = payoff;
  }
}

template< int NUM_THREADS_PER_BLOCK >
__global__ __launch_bounds__(NUM_THREADS_PER_BLOCK)
void compute_partial_sums_kernel(int num_paths, const double *__restrict cashflows, double *__restrict sums)
{
  // Shared memory to compute the final sum.
  __shared__ double lsum;

  // Each thread works on a single path.
  const int path = blockIdx.x * NUM_THREADS_PER_BLOCK + threadIdx.x;

  // Load the final sum.
  double sum = 0.0;
  if( path < num_paths )
    sum = cashflows[path];

  // Compute the sum over the block.
  if (threadIdx.x == 0)
    lsum = 0;
  __syncthreads();
  
  atomicAdd(&lsum, sum);
  __syncthreads();

  // The block leader writes the sum to GMEM.
  if( threadIdx.x == 0 )
    sums[blockIdx.x] = lsum;
}

template< int NUM_THREADS_PER_BLOCK >
__global__ __launch_bounds__(NUM_THREADS_PER_BLOCK)
void compute_final_sum_kernel(int num_paths, int num_blocks, double exp_min_r_dt, double *__restrict sums)
{
  // Shared memory to compute the final sum.
  __shared__ double lsum;

  // The sum.
  double sum = 0.0;
  for( int item = threadIdx.x ; item < num_blocks ; item += NUM_THREADS_PER_BLOCK )
    sum += sums[item];

  // Compute the sum over the block.
  if (threadIdx.x == 0) lsum = 0;
  __syncthreads();
  
  atomicAdd(&lsum, sum);
  __syncthreads();

  // The block leader writes the sum to GMEM.
  if( threadIdx.x == 0 )
  {
    sums[0] = exp_min_r_dt * lsum / (double) num_paths;
  }
}

template< typename Payoff >
static inline 
void do_run(double *h_samples,
            int num_timesteps, 
            int num_paths, 
            const Payoff &payoff, 
            double dt,
            double S0,
            double r,
            double sigma,
            double *d_samples,
            double *d_paths,
            double *d_cashflows,
            double *d_svds,
            int    *d_all_out_of_the_money,
            double *d_temp_storage,
            double *h_price)
{
  CHECK_CUDA(hipMemcpy(d_samples, h_samples, sizeof(double) * num_timesteps*num_paths, hipMemcpyHostToDevice));

  // Generate asset prices.
  const int NUM_THREADS_PER_BLOCK0 = 256;
  int grid_dim = (num_paths + NUM_THREADS_PER_BLOCK0-1) / NUM_THREADS_PER_BLOCK0;
  generate_paths_kernel<NUM_THREADS_PER_BLOCK0><<<grid_dim, NUM_THREADS_PER_BLOCK0>>>(
    num_timesteps,
    num_paths,
    payoff, 
    dt, 
    S0, 
    r, 
    sigma, 
    d_samples,
    d_paths);
  CHECK_CUDA(hipGetLastError());

  // Reset the all_out_of_the_money array.
  CHECK_CUDA(hipMemsetAsync(d_all_out_of_the_money, 0, num_timesteps*sizeof(int)));

  // Prepare the SVDs.
  const int NUM_THREADS_PER_BLOCK1 = 256;
  prepare_svd_kernel<NUM_THREADS_PER_BLOCK1><<<num_timesteps-1, NUM_THREADS_PER_BLOCK1>>>(
    num_paths,
    4, //1024,
    payoff, 
    d_paths, 
    d_all_out_of_the_money,
    d_svds);
  CHECK_CUDA(hipGetLastError());

  // The constant to discount the payoffs.
  const double exp_min_r_dt = std::exp(-r*dt);

  // Number of threads per wave at fully occupancy.
  const int num_threads_per_wave_full_occupancy = 256 * 112;

  // Enable 8B mode for SMEM.
  const int NUM_THREADS_PER_BLOCK2 = 128;

  // Update the cashflows.
  grid_dim = (num_paths + NUM_THREADS_PER_BLOCK2-1) / NUM_THREADS_PER_BLOCK2;
  double num_waves = grid_dim*NUM_THREADS_PER_BLOCK2 / (double) num_threads_per_wave_full_occupancy;

  int update_cashflow_grid = grid_dim;
  if( num_waves < 10 && num_waves - (int) num_waves < 0.6 )
    update_cashflow_grid = std::max(1, (int) num_waves) * num_threads_per_wave_full_occupancy / NUM_THREADS_PER_BLOCK2;

  // Run the main loop.
  for( int timestep = num_timesteps-2 ; timestep >= 0 ; --timestep )
  {
    // Compute beta (two kernels) for that timestep.
    compute_partial_beta_kernel<NUM_THREADS_PER_BLOCK2><<<NUM_THREADS_PER_BLOCK2, NUM_THREADS_PER_BLOCK2>>>(
      num_paths,
      payoff,
      d_svds + 16*timestep,
      d_paths + timestep*num_paths,
      d_cashflows,
      d_all_out_of_the_money + timestep,
      d_temp_storage);
    CHECK_CUDA(hipGetLastError());

    compute_final_beta_kernel<NUM_THREADS_PER_BLOCK2><<<1, NUM_THREADS_PER_BLOCK2>>>(
      d_all_out_of_the_money + timestep,
      d_temp_storage);
    CHECK_CUDA(hipGetLastError());

    update_cashflow_kernel<NUM_THREADS_PER_BLOCK2><<<update_cashflow_grid, NUM_THREADS_PER_BLOCK2>>>(
      num_paths,
      payoff,
      exp_min_r_dt,
      d_temp_storage,
      d_paths + timestep*num_paths,
      d_all_out_of_the_money + timestep,
      d_cashflows);
    CHECK_CUDA(hipGetLastError());
  }

  // Compute the final sum.
  const int NUM_THREADS_PER_BLOCK4 = 128;
  grid_dim = (num_paths + NUM_THREADS_PER_BLOCK4-1) / NUM_THREADS_PER_BLOCK4;
  
  compute_partial_sums_kernel<NUM_THREADS_PER_BLOCK4><<<grid_dim, NUM_THREADS_PER_BLOCK4>>>(
    num_paths,
    d_cashflows,
    d_temp_storage);
  CHECK_CUDA(hipGetLastError());

  compute_final_sum_kernel<NUM_THREADS_PER_BLOCK4><<<1, NUM_THREADS_PER_BLOCK4>>>(
    num_paths,
    grid_dim,
    exp_min_r_dt,
    d_temp_storage);
  CHECK_CUDA(hipGetLastError());

  // Copy the result to the host.
  CHECK_CUDA(hipMemcpy(h_price, d_temp_storage, sizeof(double), hipMemcpyDeviceToHost));
}

template< typename Payoff >
static double binomial_tree(int num_timesteps, const Payoff &payoff, double dt, double S0, double r, double sigma)
{
  double *tree = new double[num_timesteps+1];

  double u = std::exp( sigma * std::sqrt(dt));
  double d = std::exp(-sigma * std::sqrt(dt));
  double a = std::exp( r     * dt);
  
  double p = (a - d) / (u - d);
  
  double k = std::pow(d, num_timesteps);
  for( int t = 0 ; t <= num_timesteps ; ++t )
  {
    tree[t] = payoff(S0*k);
    k *= u*u;
  }

  for( int t = num_timesteps-1 ; t >= 0 ; --t )
  {
    k = std::pow(d, t);
    for( int i = 0 ; i <= t ; ++i )
    {
      double expected = std::exp(-r*dt) * (p*tree[i+1] + (1.0 - p)*tree[i]);
      double earlyex = payoff(S0*k);
      tree[i] = std::max(earlyex, expected);
      k *= u*u;
    }
  }

  double f = tree[0];
  delete[] tree;
  return f;
}

static double black_scholes_merton_put(double T, double K, double S0, double r, double sigma)
{
  double d1 = (std::log(S0 / K) + (r + 0.5*sigma*sigma)*T) / (sigma*std::sqrt(T));
  double d2 = d1 - sigma*std::sqrt(T);
  
  return K*std::exp(-r*T)*normcdf(-d2) - S0*normcdf(-d1);
}

static double black_scholes_merton_call(double T, double K, double S0, double r, double sigma)
{
  double d1 = (std::log(S0 / K) + (r + 0.5*sigma*sigma)*T) / (sigma*std::sqrt(T));
  double d2 = d1 - sigma*std::sqrt(T);
  
  return S0*normcdf(d1) - K*std::exp(-r*T)*normcdf(d2);
}

int main(int argc, char **argv)
{
  const int MAX_GRID_SIZE = 2048;
  
  // Simulation parameters.
  int num_timesteps = 100;
  int num_paths     = 32;
  int num_runs      = 1;

  // Option parameters.
  double T     = 1.00;
  double K     = 4.00;
  double S0    = 3.60;
  double r     = 0.06;
  double sigma = 0.20;

  // Bool do we price a put or a call.
  bool price_put = true;
  
  // Read command-line options.
  for( int i = 1 ; i < argc ; ++i )
  {
    if( !strcmp(argv[i], "-timesteps") )
      num_timesteps = strtol(argv[++i], NULL, 10);
    else if( !strcmp(argv[i], "-paths") )
      num_paths = strtol(argv[++i], NULL, 10);
    else if( !strcmp(argv[i], "-runs") )
      num_runs = strtol(argv[++i], NULL, 10);
    else if( !strcmp(argv[i], "-T") )
      T = strtod(argv[++i], NULL);
    else if( !strcmp(argv[i], "-S0") )
      S0 = strtod(argv[++i], NULL);
    else if( !strcmp(argv[i], "-K") )
      K = strtod(argv[++i], NULL);
    else if( !strcmp(argv[i], "-r") )
      r = strtod(argv[++i], NULL);
    else if( !strcmp(argv[i], "-sigma") )
      sigma = strtod(argv[++i], NULL);
    else if( !strcmp(argv[i], "-call") )
      price_put = false;
    else
    {
      fprintf(stderr, "Unknown option %s. Aborting!!!\n", argv[i]);
      exit(1);
    }
  }

  // Print the arguments.
  printf("==============\n");
  printf("Num Timesteps         : %d\n",  num_timesteps);
  printf("Num Paths             : %dK\n", num_paths);
  printf("Num Runs              : %d\n",  num_runs);
  printf("T                     : %lf\n", T);
  printf("S0                    : %lf\n", S0);
  printf("K                     : %lf\n", K);
  printf("r                     : %lf\n", r);
  printf("sigma                 : %lf\n", sigma);
  printf("Option Type           : American %s\n",  price_put ? "Put" : "Call");

  // We want x1024 paths.
  num_paths *= 1024;

  // A timestep.
  double dt = T / num_timesteps;

  // Generate random samples on a host
  std::default_random_engine rng;
  std::normal_distribution<double> norm_dist(0.0, 1.0);

  double *h_samples = (double*) malloc (num_timesteps*num_paths*sizeof(double));

  // Memory on the GPU to store normally distributed random numbers.
  double *d_samples = NULL;
  CHECK_CUDA(hipMalloc((void**) &d_samples, num_timesteps*num_paths*sizeof(double)));

  // Memory on the GPU to store the asset price along the paths. The last column contains the discounted payoffs.
  double *d_paths = NULL;
  CHECK_CUDA(hipMalloc((void**) &d_paths, num_timesteps*num_paths*sizeof(double)));

  // The discounted payoffs are the last column.
  double *d_cashflows = d_paths + (num_timesteps-1)*num_paths;

  // Storage to keep intermediate SVD matrices.
  double *d_svds = NULL;
  CHECK_CUDA(hipMalloc((void**) &d_svds, 16*num_timesteps*sizeof(double)));

  // Memory on the GPU to flag timesteps where no path is in the money.
  int *d_all_out_of_the_money = NULL;
  CHECK_CUDA(hipMalloc((void**) &d_all_out_of_the_money, num_timesteps*sizeof(int)));

  // Memory on the GPU to compute the reductions (beta and the option price).
  int max_temp_storage = 4*MAX_GRID_SIZE;
  double *d_temp_storage = NULL;
  CHECK_CUDA(hipMalloc((void**) &d_temp_storage, max_temp_storage*sizeof(double)));

  // The price on the host.
  double h_price;

  // time the do_run function
  float total_elapsed_time = 0;

  for( int run = 0; run < num_runs; ++run )
  {
    for (int i = 0; i < num_timesteps*num_paths; ++i)
      h_samples[i] = norm_dist(rng);
      
    auto start = std::chrono::high_resolution_clock::now();
    if( price_put )
      do_run(h_samples,
             num_timesteps, 
             num_paths, 
             PayoffPut(K), 
             dt,
             S0,
             r,
             sigma,
             d_samples,
             d_paths,
             d_cashflows,
             d_svds,
             d_all_out_of_the_money,
             d_temp_storage,
             &h_price);
    else
      do_run(h_samples,
             num_timesteps, 
             num_paths, 
             PayoffCall(K), 
             dt,
             S0,
             r,
             sigma,
             d_samples,
             d_paths,
             d_cashflows,
             d_svds,
             d_all_out_of_the_money,
             d_temp_storage,
             &h_price);

    auto end = std::chrono::high_resolution_clock::now();
    const float elapsed_time =
       std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    total_elapsed_time += elapsed_time;
  }

  printf("==============\n");
  printf("GPU Longstaff-Schwartz: %.8lf\n", h_price);
  
  double price = 0.0;

  if( price_put )
    price = binomial_tree(num_timesteps, PayoffPut(K), dt, S0, r, sigma);
  else
    price = binomial_tree(num_timesteps, PayoffCall(K), dt, S0, r, sigma);

  printf("Binonmial             : %.8lf\n", price);
  
  if( price_put )
    price = black_scholes_merton_put(T, K, S0, r, sigma);
  else
    price = black_scholes_merton_call(T, K, S0, r, sigma);

  printf("European Price        : %.8lf\n", price);

  printf("==============\n");

  printf("elapsed time for each run         : %.3fms\n", total_elapsed_time / num_runs);
  printf("==============\n");

  // Release memory
  free(h_samples);
  CHECK_CUDA(hipFree(d_temp_storage));
  CHECK_CUDA(hipFree(d_all_out_of_the_money));
  CHECK_CUDA(hipFree(d_svds));
  CHECK_CUDA(hipFree(d_paths));
  CHECK_CUDA(hipFree(d_samples));

  return 0;
}
